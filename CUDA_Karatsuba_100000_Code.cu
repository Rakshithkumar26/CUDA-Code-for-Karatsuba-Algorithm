#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include<hip/hip_runtime.h>

// Function to calculate the number of digits in a number
__device__ int numDigits(long long n) {
	int count = 0;
	while (n != 0) {
		n /= 10;
		count++;
	}
	return count;
}

__device__ int customMax(int a, int b) {
    return (a > b) ? a : b;
}

__global__ void multiplication(long long *d_a, long long *d_b, long long *d_c, int len){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<len){
       int x=d_a[tid];
        int y=d_b[tid];
        if (x < 10 || y < 10) {
			d_c[tid] = x * y;
		}else{
			// Calculate the number of digits in the two numbers and divide by 2
			int n = customMax(numDigits(x), numDigits(y));
			int n2 = (n / 2);

			// Split the numbers into two parts

			long long x_h = x / (long long)pow(10, n2);
			long long x_l = x % (long long)pow(10, n2);
			long long y_h = y / (long long)pow(10, n2);
			long long y_l = y % (long long)pow(10, n2);

			// Recursively calculate the three products
			long long high_prod = x_h * y_h;
			long long low_prod = x_l * y_l;
			long long inter_prod = ((x_h + x_l) * (y_h + y_l));
			long long subtract = inter_prod - high_prod - low_prod;

			// Calculate and return the final result
			d_c[tid] = (high_prod * (long long)pow(10, 2 * n2)) + (subtract * (long long)pow(10, n2)) + low_prod;
		}
    }
}

int main(int argc, char const *argv[]){

    long long *d_a, *d_b, *d_c;
    long long C[100000];
    float import_time;
    float exe_time;
    float export_time;

    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    hipEventRecord(start1, 0);

    FILE *file1 = fopen("X_100000.txt", "r");
	if (file1 == NULL) {
        printf("Failed to open the file for reading.\n");
        return 1;
    }
    long long A[100000];
    int num_elements_A = 0;

    // Read integers from the file and store them in an array
    while (fscanf(file1, "%lld", &A[num_elements_A]) != EOF) {
        num_elements_A++;
    }

    int num_of_elements = num_elements_A;

    FILE *file2 = fopen("Y_100000.txt", "r");
	if (file2 == NULL) {
        printf("Failed to open the file for reading.\n");
        return 1;
    }
    long long B[100000];
    int num_elements_B = 0;

    // Read integers from the file and store them in an array
    while (fscanf(file2, "%lld", &B[num_elements_B]) != EOF) {
        num_elements_B++;
    }

    hipEventRecord(stop1, 0);
    hipEventElapsedTime(&import_time, start1, stop1);

    hipMalloc((void **)&d_a, num_of_elements*sizeof(long long int));
    hipMalloc((void **)&d_b, num_of_elements*sizeof(long long int));
    hipMalloc((void **)&d_c, num_of_elements*sizeof(long long int));

    hipMemcpy(d_a, A, num_of_elements*sizeof(long long int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, num_of_elements*sizeof(long long int), hipMemcpyHostToDevice);

    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    hipEventRecord(start2, 0);

    int blockSize = 256;  // You can adjust this based on your requirements
    int numBlocks = (num_of_elements + blockSize - 1) / blockSize;
    multiplication<<<numBlocks, blockSize>>>(d_a, d_b, d_c, num_of_elements);

    hipMemcpy(C, d_c, num_of_elements*sizeof(long long int), hipMemcpyDeviceToHost);

    hipEventRecord(stop2, 0);
    hipEventElapsedTime(&exe_time, start2, stop2);

    hipEvent_t start3, stop3;
    hipEventCreate(&start3);
    hipEventCreate(&stop3);

    hipEventRecord(start3, 0);

    FILE *file3 = fopen("cudaproduct_100000.txt", "w");
    if (file3 == NULL) {
        printf("Failed to open the file for writing.\n");
        return 1;
    }
    for (int i = 0; i < num_of_elements; i++) {
        fprintf(file3, "%llu\n", C[i]);
    }

    hipEventRecord(stop3, 0);
    hipEventElapsedTime(&export_time, start3, stop3);

    printf("Time taken for importing dataset is : %fms.\n", import_time);
    printf("Time taken for Execution is : %fms.\n", exe_time);
    printf("Time taken for exporting dataset is : %fms.\n", export_time);
    printf("Product values are available in file cudaproduct_100000.txt\n");
	fclose(file1);
	fclose(file2);
	fclose(file3);
	return 0;
}